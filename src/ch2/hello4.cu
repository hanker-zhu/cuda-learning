#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU from block (%d, %d, %d) and thread (%d, %d, %d)\n",
           blockIdx.x, blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z);
    
    // sleep random time to simulate work
    unsigned long long int sleep_time = 1000000 * 1000;
    printf("Sleeping for %llu seconds\n", sleep_time);
    unsigned long long int start = clock64();
    while (clock64() - start < sleep_time) {
        // Busy wait for the specified time 
    }
    printf("Finished sleeping in block (%d, %d, %d) and thread (%d, %d, %d)  start clock is (%llu)\n",
           blockIdx.x, blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z,
            start
        );
}

int main(void) {
    dim3 grid(1, 2, 1);
    dim3 block(1, 1, 3);

    printf("Launching kernel with %d blocks and %d threads per block\n", 
        grid.x * grid.y * grid.z, block.x * block.y * block.z);
    printf("Hello from CPU\n");

    // Launch the kernel
    hello_from_gpu<<<grid, block>>>();

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));


    printf("Hello from CPU\n");
    return 0;
}