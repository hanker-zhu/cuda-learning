#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU from block %d, thread %d!\n", blockIdx.x, threadIdx.x);
    // Note: blockIdx.x is the block index in the grid, and threadIdx.x
}

int main(void) {
    hello_from_gpu<<<4, 4>>>();
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    printf("Hello from CPU\n");
    return 0;
}