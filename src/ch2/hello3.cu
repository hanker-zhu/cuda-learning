#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU from block %d, thread %d!\n", blockIdx.x, threadIdx.x);
    // Note: blockIdx.x is the block index in the grid, and threadIdx.x
}

int main(void) {
    hello_from_gpu<<<3, 4>>>();
    printf("Hello from CPU\n");
    return 0;
}