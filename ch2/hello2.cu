#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main(void) {
    hello_from_gpu<<<1, 1>>>();
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    printf("Hello from CPU\n");
    return 0;
}